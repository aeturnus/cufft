
#include <hip/hip_runtime.h>
#ifdef GPU_EN

#include <dsp/signal.h>
#include <cuda.h>
#include <cuComplex.h>

/**
 * multiply-accumulate for each slide
 *
 * Since processing y[n] has a sequential dependency
 * on h[n'], h[n'+1], h[n'+2] etc due to the MAC summation,
 * process one MAC across all shifts at once. Thus, for y of size N,
 * we launch N threads.
 *
 * @param[in]   x
 * @param[in]   h
 * @param[in]   y
 * @param[in]   xLen
 * @param[in]   hLen
 * @param[in]   yLen
 * @param[in]   hI      position in h[m]
 * @param[in]   offset  current offset in our processing,
 *                      since there's a block dimension limit
 */
// TODO: we can optimize the parameters: what do we really need?
__global__
void mac_slide( const cuFloatComplex * x, const cuFloatComplex * h, cuFloatComplex *y,
                int                 xLen, int                 hLen, int          yLen,
                int hI, int offset)
{
    // TODO: have calculations for offset. Thus we're limited to
    //       1024*1024 threads for the time being
    int n = blockIdx.x * blockDim.x + threadIdx.x; // TODO: fix the calculation
    int xI = n - hI;
    cuFloatComplex sum  = y[n];  // pick up the last sum
    cuFloatComplex xVal = make_cuFloatComplex(0.0f,0.0f);
    if( 0 <= xI && xI < xLen )
    {
        
    }
    cuFloatComplex hVal = h[hI]; // TODO: optimization: provide hVal from host
                                 //       don't need to memcpy h[n]
    cuFloatComplex product = cuCmulf(xVal, hVal);   // multiply
    y[n] = cuCaddf(sum, product);                   // accumulate
}

void complex2cuFloatComplex( int len, const std::complex<float> cpp[], cuFloatComplex cu[] )
{
    for(int i = len-1; i >= 0; --i)
    {
        cu[i] = make_cuFloatComplex( cpp[i].real(), cpp[i].imag() );
    }
}

void cuFloatComplex2complex( int len, const cuFloatComplex cu[], std::complex<float> cpp[])
{
    for(int i = len-1; i >= 0; --i)
    {
        cpp[i] = std::complex<float>(cuCrealf(cu[i]),cuCimagf(cu[i]));
    }
}

using namespace dsp;
Signal Signal::convolve( const Signal & x, const Signal & h )
{
    int xLen    = x.m_length;
    int hLen    = h.m_length;
    int yLen    = xLen + hLen - 1;

    const std::complex<float> * x_buffer = x.m_samples;
    const std::complex<float> * h_buffer = h.m_samples;
    std::complex<float> * y_buffer = new std::complex<float>[yLen];

    cuFloatComplex * h_x, * h_h, * h_y, * d_x, * d_h, * d_y;
    h_x = new cuFloatComplex[xLen];
    h_h = new cuFloatComplex[hLen];
    h_y = new cuFloatComplex[yLen];
    for(int i = yLen-1; i >= 0; --i)
    {
        y_buffer[i] = std::complex<float>(0,0);    // zero-init
    }
    // convert std::complex to cuFloatComplex
    complex2cuFloatComplex( xLen, x_buffer, h_x );
    complex2cuFloatComplex( hLen, h_buffer, h_h );
    
    cudaMalloc(&d_x, sizeof(cuFloatComplex)*xLen);
    cudaMalloc(&d_h, sizeof(cuFloatComplex)*hLen);
    cudaMalloc(&d_y, sizeof(cuFloatComplex)*yLen);

    cudaMemcpy(d_x, h_x, sizeof(cuFloatComplex)*xLen, cudaMemcpyHostToDevice);
    cudaMemcpy(d_h, h_h, sizeof(cuFloatComplex)*hLen, cudaMemcpyHostToDevice);
    // launch our things in parallel,
    // then lockstep the MAC
    for(int hI = 0; hI < hLen; ++hI)
    {
        mac_slide<<<yLen,yLen/1024>>>( d_x,  d_h,  d_y,
                                       xLen, hLen, yLen,
                                       hI, 0 );
    }
    cudaMemcpy(d_y, h_y, sizeof(cuFloatComplex)*hLen, cudaMemcpyDeviceToHost);

    cuFloatComplex2complex( yLen, h_y, y_buffer );
    Signal y;
    y.m_samples  = y_buffer;
    y.m_length   = yLen;
    y.m_position = x.m_position - hLen + 1; // since x stays in place
    
    cudaFree(d_x);
    cudaFree(d_h);
    cudaFree(d_y);
    delete[] h_x;
    delete[] h_h;
    delete[] h_y;
    return y;
}
#endif
